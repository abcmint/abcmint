#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <sys/time.h>
#include <inttypes.h>

#define K PARK
#define NUM_THREADS (1 << (N-K))
#define BLOCK_DIM (NUM_THREADS > 128 ? 128 : NUM_THREADS)
#define GRID_DIM (NUM_THREADS/BLOCK_DIM)
#define PRINT_SOL(X)
#define LOG(level, f_, ...)
#define COEF(I,J) ((((J)*((J)-1))>>1) + (I))

template <class Type>
class cudaData
{
public:
	cudaData(size_t len_host, size_t len_dev=0);
	~cudaData();

	size_t size_host();
	size_t size_dev();
	void clear();
	void write(size_t off_src=0, size_t size=0, size_t off_des=0);		
	void read(size_t off_src=0, size_t size=0, size_t off_des=0);

	Type *host;
	Type *dev;

private:
	size_t sz_host;
	size_t sz_dev;
};


template <class Type>
cudaData<Type>::cudaData(size_t len_host, size_t len_dev)
{
	if(len_dev == 0) len_dev = len_host;

	sz_host = len_host * sizeof(Type);
	sz_dev  = len_dev * sizeof(Type);

	host = (Type*) malloc(sz_host);	

	hipMalloc((void**) &dev, sz_dev);
}

template <class Type>
cudaData<Type>::~cudaData()
{
    if(host)
        free(host);
    if(dev)
        hipFree(dev);
}

template <class Type>
size_t cudaData<Type>::size_host()
{
	return sz_host;
}

template <class Type>
size_t cudaData<Type>::size_dev()
{
	return sz_dev;
}

template <class Type>
void cudaData<Type>::clear()
{
	memset(host, 0, sz_host);
}

template <class Type>
void cudaData<Type>::write(size_t off_src, size_t size, size_t off_des)
{
	if(size == 0) size = (sz_host <= sz_dev) ? sz_host : sz_dev;

	hipMemcpy(&dev[off_src], &host[off_des], size, hipMemcpyHostToDevice);
}

template <class Type>
void cudaData<Type>::read(size_t off_src, size_t size, size_t off_des)
{
	if(size == 0) size = (sz_host <= sz_dev) ? sz_host : sz_dev;

	hipMemcpy(&host[off_src], &dev[off_des], size, hipMemcpyDeviceToHost);
}

#ifndef HAVE_CNT
static const int MultiplyDeBruijnBitPosition[32] = 
{
    0, 1, 28, 2, 29, 14, 24, 3, 30, 22, 20, 15, 25, 17, 4, 8, 
      31, 27, 13, 23, 21, 19, 16, 7, 26, 12, 18, 6, 11, 5, 10, 9
};

static uint32_t cnt0(uint32_t v)
{
  return MultiplyDeBruijnBitPosition[((uint32_t)((v & -v) * 0x077CB531U)) >> 27];
}
#define HAVE_CNT
#endif


uint32_t* check_thread(uint32_t *deg2, uint32_t *deg1, uint32_t thread, uint32_t N)
{
  uint32_t rounds;
  uint32_t x = 0;
  uint32_t y = 0;
  uint32_t z = 0;
  uint32_t tmp = 0;
  uint32_t count = 0;

  uint32_t diff[K];

  uint32_t* result = (uint32_t*) malloc( (1 << K) * sizeof(uint32_t) );

  diff[0] = deg1[0 * NUM_THREADS + thread];

  for (int i = 1; i < K; i++)
  {
    diff[i] = deg1[i * NUM_THREADS + thread] ^ deg2[COEF(i-1, i)];
  }

  uint32_t res = deg1[K * NUM_THREADS + thread];

  for( rounds = 1; rounds < (1 << K); rounds += 1)
  {
    tmp = (rounds & (rounds-1));
    y = rounds ^ tmp;
    x ^= y;
    z = tmp ^ (tmp & (tmp-1));

    uint32_t y_pos = cnt0(y);
    uint32_t z_pos = cnt0(z);

    if (z_pos > y_pos)
      diff[y_pos] ^= deg2[COEF(y_pos, z_pos)];

    res ^= diff[y_pos];
    if( res == 0 ) result[ count++ ] = x;
  }

  result[ count ] = 0;

  return result;
}


void deg0_coefs(uint32_t *deg2, uint32_t *deg1, uint32_t *result, uint32_t N)
{
	uint32_t rounds;
	uint32_t x = 0;
	uint32_t y = 0;
	uint32_t z = 0;
	uint32_t tmp = 0;

  uint32_t diff[N-K];

	diff[0] = deg1[0];

  for (int i = 1; i < (N-K); i++)
  {
    diff[i] = deg1[i] ^ deg2[COEF(i-1, i)];
  }

	uint32_t res = deg1[N-K];

	result[0] = res;

	for (rounds = 1; rounds < (1 << (N-K)); rounds += 1)
	{
		tmp = (rounds & (rounds-1));
		y = rounds ^ tmp;
		x ^= y;
		z = tmp ^ (tmp & (tmp-1));

    uint32_t y_pos = cnt0(y);
    uint32_t z_pos = cnt0(z);

    if (z_pos > y_pos)
      diff[y_pos] ^= deg2[COEF(y_pos, z_pos)];

    res ^= diff[y_pos];
    result[ x ] = res;
    tmp = (y_pos * (y_pos-1)) >> 1;
	}
}

void deg1_coefs(uint32_t *deg1, uint32_t *result, uint32_t N)
{
	uint32_t x = 0;
	uint32_t y = 0;
	uint32_t res = deg1[N-K];

	result[0] = res;

	for(uint32_t rounds = 1; rounds < (1 << (N-K)); rounds += 1)
	{
		y = rounds ^ (rounds & (rounds-1));
		x ^= y;

    res ^= deg1[cnt0(y)];
    result[ x ] = res;
	}
}

void partial_eval(uint32_t *sys, uint32_t *deg1, uint32_t N)
{
	uint32_t deg1_sys[(N-K)+1];
	uint32_t deg2_sys[COEF(N-K, N-K)+1];
	uint64_t pos = 0;

	// deg2 part
	for(uint32_t i = 0; i < K; i++)
	{
    for(uint32_t j = 0; j <= (N-K); j++)
    {
      deg1_sys[j] = sys[COEF(0, j + K) + i];
    }

		deg1_coefs(deg1_sys, &deg1[pos], N);
		pos += (1 << (N-K));
	}

	// deg1 part
  for(uint32_t j = 1; j <= (N-K); j++)
  {
    for(uint32_t i = 0; i <= j; i++)
    {
      deg2_sys[COEF(i, j)] = sys[COEF(i + K, j + K)];
    }
  }

  deg0_coefs(deg2_sys, deg2_sys + COEF(0, N-K), &deg1[pos], N);
}



uint32_t check_sol(uint32_t *sys, uint64_t sol, uint32_t N, uint32_t M)
{
	uint32_t i, j, pos = 0;
	uint32_t x[N], check = 0;
   
  for (uint32_t b = 0; b < M; b +=32)
  {
    uint32_t mask = (M-b) >= 32 ? 0xffffffff : ((1 << (M-b))-1);

    for (i = 0; i < N; i++)
      x[i] = ((sol >> i) & 1) ? mask : 0;

    // computing quadratic part
    for(j = 1; j < N; j++)
      for(i = 0; i < j; i++) 
        check ^= sys[pos++] & x[i] & x[j];

    // computing linear part
    for(i = 0; i < N; i++) 
      check ^= sys[pos++] & x[i];

    // constant part
    check ^= sys[pos++];
  }

	return check;
}



double get_ms_time(void) {
	struct timeval timev;

	gettimeofday(&timev, NULL);
	return (double) timev.tv_sec * 1000 + (double) timev.tv_usec / 1000;
}

extern uint32_t check_sol(uint32_t *sys, uint64_t sol, uint32_t N, uint32_t M);
extern void partial_eval(uint32_t *sys, uint32_t *deg1, uint32_t N);
extern uint32_t* check_thread(uint32_t *deg2, uint32_t *deg1, uint32_t thread, uint32_t N);

__device__ __constant__ uint32_t deg2_block[ K*(K-1)/2 ];

#include "kernel.inc"

uint32_t *pack_sys_data(uint32_t *data, uint32_t N, uint32_t M)
{
//  reduce input system - remove squares

  uint32_t num_blocks = ((M >> 5) + ((M & 31) == 0 ? 0 : 1));

  uint32_t *sys = (uint32_t*)malloc(sizeof(uint32_t)*(N*(N-1)/2 + N + 1)*num_blocks);
  uint32_t *sq0 = (uint32_t*)malloc(sizeof(uint32_t)*N*num_blocks);


  int sq_id = 0;

  int is = 0;
  int id = 0;
  
  for (int v0 = 0; v0 < N; v0++)
  {
    for (int v1 = 0; v1 <= v0; v1++)
    {
      for (uint32_t b = 0; b < M; b +=32)
      {
        uint32_t val = 0;

        for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
          val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

        if (v0 == v1)
          sq0[sq_id + N*(b >> 5)] = val;
        else
          sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val;
      }

      is += 1;

      if (v0 == v1)
        sq_id += 1;
      else
        id += 1;
    }
  }

  for (int v0 = 0; v0 < N; v0++)
  {
    for (uint32_t b = 0; b < M; b +=32)
    {
      uint32_t val = 0;

      for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
        val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

      sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val ^ sq0[v0 + N*(b >> 5)];
    }

    is += 1;
    id += 1;
  }

  {
    for (uint32_t b = 0; b < M; b +=32)
    {
      uint32_t val = 0;

      for (int j = (((M - b) >= 32) ? b + 31 : (M-1)); j >= (int)b; j--)
        val = (val << 1) | data[(N*(N-1)/2 + N + N + 1) * j + is];

      sys[(N*(N-1)/2 + N + 1) * (b >> 5) + id] = val;
    }
  }

  if(sq0)
    free(sq0);

  return sys;
}

int GetDeviceCount() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err == hipSuccess) {
      return deviceCount;
    }
    fprintf(stderr, "hipGetDeviceCount error=%d\n", err);
    return -1;
}

void SetDevice(int device)
{
    hipSetDevice(device);
}

uint64_t GPUSearchSolution(uint32_t* coefficients, unsigned int number_of_variables, 
                           unsigned int number_of_equations)
{
  uint64_t res = 0;

  uint32_t N = number_of_variables;
  uint32_t M = number_of_equations;


  if (N <= K)
  {
    fprintf(stderr, "N must be larger than K!\n");
    exit(-1);
  }

  uint32_t *sys = pack_sys_data(coefficients, N, M);


  cudaData<uint32_t> deg1((K + 1) * NUM_THREADS);

  partial_eval(sys, deg1.host, N);


  // initialize constant memory space for the quadratic part
  hipMemcpyToSymbol(HIP_SYMBOL(deg2_block), sys, sizeof(uint32_t) * K*(K-1)/2);

  // initialize global memory space for the linear parts
  deg1.write();

  // initialize global memory space for the results of each threads
  cudaData<uint32_t> result(NUM_THREADS);



  guess<<<GRID_DIM, BLOCK_DIM>>>(deg1.dev, result.dev, NUM_THREADS);



  result.read();



  int32_t ans; 

  for(uint64_t i = 0; i < NUM_THREADS; i++)
  {
    ans = result.host[i];

    if(ans)
    {

      if (ans & 0x80000000) // more than one solution 
      {
        uint32_t * sols = check_thread(sys, deg1.host, i, N);


        for(uint32_t j = 0; sols[j]; j++)
        {
          if (check_sol(sys, (i << K) | sols[j], N, M) == 1)
          {
            PRINT_SOL((i << K) | sols[j]);

            res = (i << K) | sols[j];
            free(sols);
            goto end;
           
          }
        }


        free(sols);
      }
      else // only one solution
      {
        if (check_sol(sys, (i << K) | ans, N, M) == 0)
        {
          PRINT_SOL((i << K) | ans);

          res = (i << K) | ans;

          goto end;
        }
      }
    }

    if(deg1.host[K * NUM_THREADS + i] == 0)
    {
      if (check_sol(sys, (i << K) | 0, N, M) == 0)
      {
        PRINT_SOL(i << K);

        res = (i << K);

        goto end;
      }
    }
  }

end:

    if(sys)
        free(sys);

  return res;
}
